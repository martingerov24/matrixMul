#include "hip/hip_runtime.h"
#include "../external/CudaMatrix.h"



__global__
void Kernel(int32_t* matrix1, int32_t * matrix2,int32_t* result, int32_t m_x, int32_t other_y)
{
	if ( m_x != other_y
		|| m_x == 0 
		|| other_y == 0)
	{
		return;
	}

	int resultIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if (resultIndex >= m_x * other_y
		|| resultIndex < 0)
	{
		return;
	}

	__shared__ int32_t s_matrix[100*100];
	//__shared__ int32_t s_matrix2[100 * 100];
	s_matrix[resultIndex] = matrix1[resultIndex];
	//s_matrix2[resultIndex] = matrix2[resultIndex];
	__syncthreads();
	uint8_t A = resultIndex / m_x;
	uint8_t B = resultIndex - A * m_x;

	int32_t res = 0;

	for (int i = 0; i < m_x; i++)
	{
		res += s_matrix[A * other_y+i] * matrix2[B + m_x * i];
	}

	result[resultIndex] = res;
}

__global__
void DivideAndConquer(int32_t* matrix1, int32_t* matrix2, int32_t* result, const int32_t m_x, const int32_t other_y)
{
	int resultIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if (resultIndex >= m_x * other_y
		|| resultIndex < 0)
	{
		return;
	}

	int32_t wBord; 
	uint8_t A = resultIndex / m_x;
	uint8_t B = resultIndex - A * m_x;
	wBord = A / BORDER_SIZE + B / BORDER_SIZE;

	__shared__ int32_t A_matrixRow[36];
	__shared__ int32_t B_matrixCol[36];
	B_matrixCol[resultIndex] = matrix2[resultIndex];
	__shared__ int32_t Result[36];
	__syncthreads();


	for (int i = 0; i < m_x; i++)
	{
		A_matrixRow[resultIndex] = matrix1[resultIndex];

	}
	//__shared__ int32_t s_matrix2[100 * 100];
	//s_matrix2[resultIndex] = matrix2[resultIndex];

	result[resultIndex] = res;
}


__host__
void Cuda::MatrixMultiplication(hipStream_t& providedStream, int32_t sharedMem)
{

	int8_t sizeOfBlock(((x * y + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK));
	NVPROF_SCOPE("kernel");
	Kernel << <sizeOfBlock, THREADS_PER_BLOCK, 0, providedStream >> > (m_matrix1, m_matrix2, resultM, x, y);
	auto status = hipGetLastError();
}