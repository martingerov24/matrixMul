#include "hip/hip_runtime.h"
#include "../external/CudaMatrix.h"

__device__
union intToDim
{
	struct dimentionsOfMatrix {
		int32_t m_x : 8;
		int32_t m_y : 8;
		int32_t other_x: 8;
		int32_t other_y : 8;
	};

	int32_t value;
};

__global__
void Kernel(int32_t* matrix1, int32_t * matrix2, int32_t dim)
{
	
}

__host__
void Cuda::MatrixMultiplication(hipStream_t& providedStream, intToDim values)
{
	dim3 sizeOfBlock(((values.dim.m_x + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK), values.dim.m_y);
	int32_t v = values.integer;
	Kernel << <sizeOfBlock, THREADS_PER_BLOCK, 0, providedStream >> > (m_matrix1, m_matrix2, v);
	auto status = hipGetLastError();
}