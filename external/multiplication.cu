#include "hip/hip_runtime.h"
#include "../external/CudaMatrix.h"

__global__
void Kernel(int32_t* matrix1, int32_t * matrix2,int32_t* result, uint32_t dim)
{
	const uint8_t other_y = dim & 0xff;
	const uint8_t m_x = dim >> 24;

	if ( m_x != other_y
		|| m_x == 0 
		|| other_y == 0)
	{
		return;
	}

	int resultIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if (resultIndex >= m_x * other_y
		|| resultIndex < 0)
	{
		return;
	}

	__shared__ int32_t s_matrix[100*100];
	//__shared__ int32_t s_matrix2[100 * 100];
	s_matrix[resultIndex] = matrix1[resultIndex];
	//s_matrix2[resultIndex] = matrix2[resultIndex];
	__syncthreads();
	uint8_t A = resultIndex / m_x;
	uint8_t B = resultIndex - A * m_x;

	int32_t res = 0;

	for (int i = 0; i < m_x; i++)
	{
		res += s_matrix[A * other_y+i] * matrix2[B + m_x * i];
	}

	result[resultIndex] = res;
}

__host__
void Cuda::MatrixMultiplication(hipStream_t& providedStream, uint32_t values)
{
	uint8_t other_y = values & 0xff;
	uint8_t m_x = values >> 24;
	int8_t sizeOfBlock(((m_x * other_y + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK));
	NVPROF_SCOPE("kernel");
	Kernel << <sizeOfBlock, THREADS_PER_BLOCK, 0, providedStream >> > (m_matrix1, m_matrix2, resultM, values);
	auto status = hipGetLastError();
}