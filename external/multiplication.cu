#include "hip/hip_runtime.h"
#include "../external/CudaMatrix.h"



__global__
void Kernel(int32_t* matrix1, int32_t * matrix2,int32_t* result, int32_t m_x, int32_t other_y)
{
	if ( m_x != other_y
		|| m_x == 0 
		|| other_y == 0)
	{
		return;
	}

	int resultIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if (resultIndex >= m_x * other_y
		|| resultIndex < 0)
	{
		return;
	}

	__shared__ int32_t s_matrix[100*100];
	//__shared__ int32_t s_matrix2[100 * 100];
	s_matrix[resultIndex] = matrix1[resultIndex];
	//s_matrix2[resultIndex] = matrix2[resultIndex];
	__syncthreads();
	uint8_t A = resultIndex / m_x;
	uint8_t B = resultIndex - A * m_x;

	int32_t res = 0;

	for (int i = 0; i < m_x; i++)
	{
		res += s_matrix[A * other_y+i] * matrix2[B + m_x * i];
	}

	result[resultIndex] = res;
}

__global__
void DivideAndConquer(int32_t* matrix1, int32_t* matrix2, int32_t* result, const int32_t m_x, const int32_t other_y)
{
	int resultIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if (resultIndex >= m_x 
		|| resultIndex < 0)
	{
		return;
	}

	int32_t wBord; 
	uint8_t A = resultIndex / m_x;
	uint8_t B = resultIndex - A * m_x;
	wBord = A / BORDER_SIZE + B / BORDER_SIZE;
	
	__shared__ int32_t A_matrixRow[BORDER_SIZE * 6]; // read 2 rows of the matrix
	__shared__ int32_t B_matrixCol[BORDER_SIZE * 6]; // read the 2 cols of the matrix

	A_matrixRow[resultIndex] = matrix1[resultIndex];
	__syncwarp();									// this might make a problem
		for (int i = 0; i < m_x ; i+=BORDER_SIZE)
		{
#pragma unroll
			for (int j = 0; j < BORDER_SIZE; j++)
			{
				B_matrixCol[j] = matrix2[B + m_x * i + j];
				__syncwarp();
			}
			int32_t m[7] =
			{
				(A_matrixRow[0] +  A_matrixRow[3]) *(B_matrixCol[0] + B_matrixCol[3]),
				(A_matrixRow[1] +  A_matrixRow[3]) *(B_matrixCol[0]),
				(A_matrixRow[0])* (B_matrixCol[2]  - B_matrixCol[3]),
				(A_matrixRow[3])* (B_matrixCol[1]  - B_matrixCol[0]),
				(A_matrixRow[0] +  A_matrixRow[2]) * B_matrixCol[3],
				(A_matrixRow[1] -  A_matrixRow[1]) *(B_matrixCol[0] + B_matrixCol[2]),
				(A_matrixRow[2] -  A_matrixRow[3]) *(B_matrixCol[0] + B_matrixCol[2]),
			};
			result[0] += m[0] + m[3] - m[4] + m[6];
			result[1] += m[2] + m[4];
			result[2] += m[1] + m[3];
			result[3] += m[0] - m[1] + m[2] + m[5];

		}
	//__shared__ int32_t s_matrix2[100 * 100];
	//s_matrix2[resultIndex] = matrix2[resultIndex];

}


__host__
void Cuda::MatrixMultiplication(hipStream_t& providedStream, int32_t sharedMem)
{

	int8_t sizeOfBlock(((x * y + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK));
	NVPROF_SCOPE("kernel");
	Kernel << <sizeOfBlock, THREADS_PER_BLOCK, 0, providedStream >> > (m_matrix1, m_matrix2, resultM, x, y);
	auto status = hipGetLastError();
}

//__host__
//void Cuda::DivideAndConquer(hipStream_t& providedStream)
//{
//	int8_t sizeOfBlock(((x + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK));
//	NVPROF_SCOPE("kernel");
//	DivideAndConquer<< <sizeOfBlock, THREADS_PER_BLOCK, 0, providedStream >> > (m_matrix1, m_matrix2, resultM, x, y);
//	auto status = hipGetLastError();
//}